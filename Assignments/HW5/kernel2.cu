#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__device__ int mandel(float c_re, float c_im, int max_iterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < max_iterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = (z_re * z_re) - (z_im * z_im);
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandel_kernel(float x0, float y0, float dx, float dy,
    int width, int height,
    int start_row, int total_rows,
    int max_iterations, int *output, size_t pitch_in_ints)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int end_row = start_row + total_rows;
    if (i < width && j >= start_row && j < end_row) {
        float x = x0 + i * dx;
        float y = y0 + j * dy;
        int index = j * pitch_in_ints + i;
        output[index] = mandel(x, y, max_iterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void host_fe(float upper_x,
             float upper_y,
             float lower_x,
             float lower_y,
             int *img,
             int res_x,
             int res_y,
             int max_iterations)
{
    float step_x = (upper_x - lower_x) / (float)res_x;
    float step_y = (upper_y - lower_y) / (float)res_y;

    int *host_output;
    size_t total_bytes = res_x * res_y * sizeof(int);
    hipHostAlloc((void**)&host_output, total_bytes, hipHostMallocDefault);

    int *output_device;
    size_t pitch_bytes;
    hipMallocPitch((void**)&output_device, &pitch_bytes, res_x * sizeof(int), res_y);
    size_t pitch_in_ints = pitch_bytes / sizeof(int);

    dim3 blockSize(16, 16);
    dim3 gridSize((res_x + blockSize.x - 1) / blockSize.x,
                  (res_y + blockSize.y - 1) / blockSize.y);

    mandel_kernel<<<gridSize, blockSize>>>(
        lower_x, lower_y, step_x, step_y,
        res_x, res_y,
        0, res_y,
        max_iterations, output_device, pitch_in_ints);

    hipDeviceSynchronize();

    hipMemcpy2D(host_output,             
        res_x * sizeof(int),      
        output_device,            
        pitch_bytes,              
        res_x * sizeof(int),      
        res_y,                    
        hipMemcpyDeviceToHost); 
    // for (int row = 0; row < res_y; ++row) {
    //     cudaMemcpy(host_output + row * res_x,
    //                 (char*)output_device + row * pitch_bytes,
    //                 res_x * sizeof(int),
    //                 cudaMemcpyDeviceToHost);
    // }

    // cudaEvent_t start, stop;
    // float memcpyTime = 0.0f;

    // cudaEventCreate(&start);
    // cudaEventCreate(&stop);
    // cudaEventRecord(start, 0);

    memcpy(img, host_output, total_bytes);

    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&memcpyTime, start, stop);

    // printf("[INFO] memcpy host_output → img took %.3f ms\n", memcpyTime);

    // cudaEventDestroy(start);
    // cudaEventDestroy(stop);

    hipHostFree(host_output);
    hipFree(output_device);
}
