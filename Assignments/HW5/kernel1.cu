#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__device__ int mandel(float c_re, float c_im, int max_iterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < max_iterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = (z_re * z_re) - (z_im * z_im);
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandel_kernel(float x0, float y0, float dx, float dy,
    int width, int height,
    int start_row, int total_rows,
    int max_iterations, int *output)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // column
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // row

    int end_row = start_row + total_rows;
    if (i < width && j >= start_row && j < end_row) {
        float x = x0 + i * dx;
        float y = y0 + j * dy;
        int index = j * width + i;
        output[index] = mandel(x, y, max_iterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void host_fe(float upper_x,
             float upper_y,
             float lower_x,
             float lower_y,
             int *img,
             int res_x,
             int res_y,
             int max_iterations)
{
    float step_x = (upper_x - lower_x) / (float)res_x;
    float step_y = (upper_y - lower_y) / (float)res_y;

    int* host_output = new int[res_x * res_y];

    int *output_device;
    size_t total_size = res_x * res_y * sizeof(int);
    hipMalloc(&output_device, total_size);

    dim3 blockSize(16, 16);
    dim3 gridSize((res_x + blockSize.x - 1) / blockSize.x,
                  (res_y + blockSize.y - 1) / blockSize.y);

    mandel_kernel<<<gridSize, blockSize>>>(
        lower_x, lower_y, step_x, step_y,
        res_x, res_y,
        0, res_y,
        max_iterations, output_device);

    hipDeviceSynchronize();
    hipMemcpy(host_output, output_device, total_size, hipMemcpyDeviceToHost);
    memcpy(img, host_output, res_x * res_y * sizeof(int));

    delete[] host_output;
    hipFree(output_device);
}
